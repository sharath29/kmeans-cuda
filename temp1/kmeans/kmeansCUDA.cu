
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

clock_t t;
double gpu_time_used;

#define I(row, col, ncols) (row * ncols + col)

#define CUDA_CALL(x) {if((x) != hipSuccess){ \
  printf("CUDA error at %s:%d\n",__FILE__,__LINE__); \
  printf("  %s\n", hipGetErrorString(hipGetLastError())); \
  exit(EXIT_FAILURE);}}

__global__ void get_dst(float *dst, float *x, float *y,
			float *mu_x, float *mu_y){
  int i = blockIdx.x;
  int j = threadIdx.x;

  dst[I(i, j, blockDim.x)] = (x[i] - mu_x[j]) * (x[i] - mu_x[j]);
  dst[I(i, j, blockDim.x)] += (y[i] - mu_y[j]) * (y[i] - mu_y[j]);
}

__global__ void regroup(int *group, float *dst, int k){
  int i = blockIdx.x;
  int j;
  float min_dst;

  min_dst = dst[I(i, 0, k)];
  group[i] = 1;

  for(j = 1; j < k; ++j){
    if(dst[I(i, j, k)] < min_dst){
      min_dst = dst[I(i, j, k)];
      group[i] = j + 1;
    }
  }
}

__global__ void clear(float *sum_x, float *sum_y, int *nx, int *ny){
  int j = threadIdx.x;

  sum_x[j] = 0;
  sum_y[j] = 0;
  nx[j] = 0;
  ny[j] = 0;
}

__global__ void recenter_step1(float *sum_x, float *sum_y, int *nx, int *ny,
			       float *x, float *y, int *group, int n){
  int i;
  int j = threadIdx.x;

  for(i = 0; i < n; ++i){
    if(group[i] == (j + 1)){
      sum_x[j] += x[i];
      sum_y[j] += y[i];
      nx[j]++;
      ny[j]++;
    }
  }
}

__global__ void recenter_step2(float *mu_x, float *mu_y, float *sum_x,
			       float *sum_y, int *nx, int *ny){
  int j = threadIdx.x;

  mu_x[j] = sum_x[j]/nx[j];
  mu_y[j] = sum_y[j]/ny[j];
}

void kmeans(int nreps, int n, int k,
            float *x_d, float *y_d, float *mu_x_d, float *mu_y_d,
            int *group_d, int *nx_d, int *ny_d,
            float *sum_x_d, float *sum_y_d, float *dst_d){
  int i;
  for(i = 0; i < nreps; ++i){
    get_dst<<<n,k>>>(dst_d, x_d, y_d, mu_x_d, mu_y_d);
    regroup<<<n,1>>>(group_d, dst_d, k);
    clear<<<1,k>>>(sum_x_d, sum_y_d, nx_d, ny_d);
    recenter_step1<<<1,k>>>(sum_x_d, sum_y_d, nx_d, ny_d, x_d, y_d, group_d, n);
    recenter_step2<<<1,k>>>(mu_x_d, mu_y_d, sum_x_d, sum_y_d, nx_d, ny_d);
  }
}

void read_data(float **x, float **y, float **mu_x, float **mu_y, int *n, int *k);
void print_results(int *group, float *mu_x, float *mu_y, int n, int k);

int main(){
  /* cpu variables */
  int n; /* number of points */
  int k; /* number of clusters */
  int *group;
  float *x = NULL, *y = NULL, *mu_x = NULL, *mu_y = NULL;

  /* gpu variables */
  int *group_d, *nx_d, *ny_d;
  float *x_d, *y_d, *mu_x_d, *mu_y_d, *sum_x_d, *sum_y_d, *dst_d;

  /* read data from files on cpu */
  read_data(&x, &y, &mu_x, &mu_y, &n, &k);

  /* allocate cpu memory */
  group = (int*) malloc(n*sizeof(int));

  /* allocate gpu memory */
  CUDA_CALL(hipMalloc((void**) &group_d,n*sizeof(int)));
  CUDA_CALL(hipMalloc((void**) &nx_d, k*sizeof(int)));
  CUDA_CALL(hipMalloc((void**) &ny_d, k*sizeof(int)));
  CUDA_CALL(hipMalloc((void**) &x_d, n*sizeof(float)));
  CUDA_CALL(hipMalloc((void**) &y_d, n*sizeof(float)));
  CUDA_CALL(hipMalloc((void**) &mu_x_d, k*sizeof(float)));
  CUDA_CALL(hipMalloc((void**) &mu_y_d, k*sizeof(float)));
  CUDA_CALL(hipMalloc((void**) &sum_x_d, k*sizeof(float)));
  CUDA_CALL(hipMalloc((void**) &sum_y_d, k*sizeof(float)));
  CUDA_CALL(hipMalloc((void**) &dst_d, n*k*sizeof(float)));

  /* write data to gpu */
  CUDA_CALL(hipMemcpy(x_d, x, n*sizeof(float), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(y_d, y, n*sizeof(float), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(mu_x_d, mu_x, k*sizeof(float), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(mu_y_d, mu_y, k*sizeof(float), hipMemcpyHostToDevice));

  t = clock();

  /* perform kmeans */
  kmeans(10, n, k, x_d, y_d, mu_x_d, mu_y_d, group_d, nx_d, ny_d, sum_x_d, sum_y_d, dst_d);

  t = clock() - t;
  gpu_time_used = ((double)t)/CLOCKS_PER_SEC;
  printf("CUDA Time taken = %lf\n",gpu_time_used);

  /* read back data from gpu */
  CUDA_CALL(hipMemcpy(group, group_d, n*sizeof(int), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(mu_x, mu_x_d, k*sizeof(float), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(mu_y, mu_y_d, k*sizeof(float), hipMemcpyDeviceToHost));

  /* print results and clean up */
  print_results(group, mu_x, mu_y, n, k);

  free(x);
  free(y);
  free(mu_x);
  free(mu_y);
  free(group);

  CUDA_CALL(hipFree(x_d));
  CUDA_CALL(hipFree(y_d));
  CUDA_CALL(hipFree(mu_x_d));
  CUDA_CALL(hipFree(mu_y_d));
  CUDA_CALL(hipFree(group_d));
  CUDA_CALL(hipFree(nx_d));
  CUDA_CALL(hipFree(ny_d));
  CUDA_CALL(hipFree(sum_x_d));
  CUDA_CALL(hipFree(sum_y_d));
  CUDA_CALL(hipFree(dst_d));

  return 0;
}

void read_data(float **x, float **y, float **mu_x, float **mu_y, int *n, int *k){
  FILE *fp;
  char buf[64];
  int i;

  *n = 0;
  fp = fopen("input/x_coordinates.txt", "r");

  while(fgets(buf, 64, fp) != NULL){
    *n += 1;
    *x = (float*) realloc(*x, (*n)*sizeof(float));
    (*x)[*n - 1] = atof(buf);
  }
  fclose(fp);

  i = 0;
  fp = fopen("input/y_coordinates.txt", "r");
  while(fgets(buf, 64, fp) != NULL){
    i += 1;
    *y = (float*) realloc(*y, i*sizeof(float));
    (*y)[i - 1] = atof(buf);
  }
  fclose(fp);

  if(i != *n){
    printf("ERROR: x.txt and y.txt must have same number of rows\n.");
    printf("That includes whitespace.\n");
    exit(EXIT_FAILURE);
  }

  *k = 0;
  fp = fopen("input/initialCluster_x_coordinates.txt", "r");
  while(fgets(buf, 64, fp) != NULL){
    *k += 1;
    *mu_x = (float*) realloc(*mu_x, (*k)*sizeof(float));
    (*mu_x)[*k - 1] = atof(buf);
  }
  fclose(fp);

  i = 0;
  fp = fopen("input/initialCluster_y_coordinates.txt", "r");
  while(fgets(buf, 64, fp) != NULL){
    i += 1;
    *mu_y = (float*) realloc(*mu_y, i*sizeof(float));
    (*mu_y)[i - 1] = atof(buf);
  }
  fclose(fp);

  if(i != *k){
    printf("ERROR: mu_x.txt and mu_y.txt must have same number of rows\n.");
    printf("That includes whitespace.\n");
    exit(EXIT_FAILURE);
  }
}

void print_results(int *group, float *mu_x, float *mu_y, int n, int k){
  FILE *fp;
  int i;

  for(i = 0; i < n; ++i)
  fp = fopen("output/cuda/cluster_members.txt", "w");
    fprintf(fp, "%d\n", group[i]);
  fclose(fp);

  fp = fopen("output/cuda/finalCluster_x_coordinates.txt", "w");
  for(i = 0; i < k; ++i)
    fprintf(fp, "%0.3f\n", mu_x[i]);
  fclose(fp);

  fp = fopen("output/cuda/finalCluster_y_coordinates.txt", "w");
  for(i = 0; i < k; ++i)
    fprintf(fp, "%0.3f\n", mu_y[i]);
  fclose(fp);
}
