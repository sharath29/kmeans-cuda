
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <algorithm>
#include <cfloat>
#include <chrono>
#include <fstream>
#include <iostream>
#include <random>
#include <sstream>
#include <stdexcept>
#include <vector>
#include <chrono>
#include <time.h>

double gpu_time_used;
#define I(row, col, ncols) (row * ncols + col)

#define CUDA_CALL(x) {if((x) != hipSuccess){ \
  printf("CUDA error at %s:%d\n",__FILE__,__LINE__); \
  printf("  %s\n", hipGetErrorString(hipGetLastError())); \
  exit(EXIT_FAILURE);}} 

__global__ void get_dst(float *dst, float *x, float *y, 
			float *mu_x, float *mu_y){
  int i = blockIdx.x;
  int j = threadIdx.x;

  dst[I(i, j, blockDim.x)] = (x[i] - mu_x[j]) * (x[i] - mu_x[j]);
  dst[I(i, j, blockDim.x)] += (y[i] - mu_y[j]) * (y[i] - mu_y[j]); 
}

__global__ void regroup(int *group, float *dst, int k){
  int i = blockIdx.x;
  int j;
  float min_dst;
  
  min_dst = dst[I(i, 0, k)];
  group[i] = 1;

  for(j = 1; j < k; ++j){
    if(dst[I(i, j, k)] < min_dst){
      min_dst = dst[I(i, j, k)];
      group[i] = j + 1;
    }
  }
}

__global__ void clear(float *sum_x, float *sum_y, int *nx, int *ny){
  int j = threadIdx.x;
  
  sum_x[j] = 0;
  sum_y[j] = 0;
  nx[j] = 0;
  ny[j] = 0;
}

__global__ void recenter_step1(float *sum_x, float *sum_y, int *nx, int *ny,
			       float *x, float *y, int *group, int n){
  int i;
  int j = threadIdx.x;

  for(i = 0; i < n; ++i){
    if(group[i] == (j + 1)){
      sum_x[j] += x[i];
      sum_y[j] += y[i];
      nx[j]++;
      ny[j]++;
    }
  }               
}

__global__ void recenter_step2(float *mu_x, float *mu_y, float *sum_x,
			       float *sum_y, int *nx, int *ny){
  int j = threadIdx.x;

  mu_x[j] = sum_x[j]/nx[j];
  mu_y[j] = sum_y[j]/ny[j];
}

void kmeans(int nreps, int n, int k,
            float *x_d, float *y_d, float *mu_x_d, float *mu_y_d,
            int *group_d, int *nx_d, int *ny_d,
            float *sum_x_d, float *sum_y_d, float *dst_d){
  int i;
  for(i = 0; i < nreps; ++i){
    get_dst<<<n,k>>>(dst_d, x_d, y_d, mu_x_d, mu_y_d);
    regroup<<<n,1>>>(group_d, dst_d, k);
    clear<<<1,k>>>(sum_x_d, sum_y_d, nx_d, ny_d);
    recenter_step1<<<1,k>>>(sum_x_d, sum_y_d, nx_d, ny_d, x_d, y_d, group_d, n);
    recenter_step2<<<1,k>>>(mu_x_d, mu_y_d, sum_x_d, sum_y_d, nx_d, ny_d);
  }
}

void read_data(float **x, float **y, float **mu_x, float **mu_y, int *n, int *k,char* arg);
void print_results(int *group, float *mu_x, float *mu_y, int n, int k,char* argv);

int main(int argc,char* argv[]){

  /* cpu variables */
  int n; /* number of points */
  int k; /* number of clusters */
  int *group;
  float *x = NULL, *y = NULL, *mu_x = NULL, *mu_y = NULL;

  /* gpu variables */
  int *group_d, *nx_d, *ny_d;
  float *x_d, *y_d, *mu_x_d, *mu_y_d, *sum_x_d, *sum_y_d, *dst_d;

  /* read data from files on cpu */
  read_data(&x, &y, &mu_x, &mu_y, &n, &k,argv[2]);

  /* allocate cpu memory */
  group = (int*) malloc(n*sizeof(int));

  /* allocate gpu memory */
  CUDA_CALL(hipMalloc((void**) &group_d,n*sizeof(int)));
  CUDA_CALL(hipMalloc((void**) &nx_d, k*sizeof(int)));
  CUDA_CALL(hipMalloc((void**) &ny_d, k*sizeof(int)));
  CUDA_CALL(hipMalloc((void**) &x_d, n*sizeof(float)));
  CUDA_CALL(hipMalloc((void**) &y_d, n*sizeof(float)));
  CUDA_CALL(hipMalloc((void**) &mu_x_d, k*sizeof(float)));
  CUDA_CALL(hipMalloc((void**) &mu_y_d, k*sizeof(float)));
  CUDA_CALL(hipMalloc((void**) &sum_x_d, k*sizeof(float)));
  CUDA_CALL(hipMalloc((void**) &sum_y_d, k*sizeof(float)));
  CUDA_CALL(hipMalloc((void**) &dst_d, n*k*sizeof(float)));

  /* write data to gpu */
  CUDA_CALL(hipMemcpy(x_d, x, n*sizeof(float), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(y_d, y, n*sizeof(float), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(mu_x_d, mu_x, k*sizeof(float), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(mu_y_d, mu_y, k*sizeof(float), hipMemcpyHostToDevice));
  /* perform kmeans */


  const auto start = std::chrono::high_resolution_clock::now();
  kmeans(100, n, k, x_d, y_d, mu_x_d, mu_y_d, group_d, nx_d, ny_d, sum_x_d, sum_y_d, dst_d);

  const auto end = std::chrono::high_resolution_clock::now();
  const auto duration =
      std::chrono::duration_cast<std::chrono::duration<float>>(end - start);
  std::cerr << "CUDA Took: " << duration.count() << "s" << " for "<<argv[3]<<" points." << std::endl;


gpu_time_used = duration.count();

  /* read back data from gpu */
  CUDA_CALL(hipMemcpy(group, group_d, n*sizeof(int), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(mu_x, mu_x_d, k*sizeof(float), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(mu_y, mu_y_d, k*sizeof(float), hipMemcpyDeviceToHost));

  /* print results and clean up */  
  print_results(group, mu_x, mu_y, n, k,argv[3]);


  free(x);
  free(y);
  free(mu_x);
  free(mu_y);
  free(group);

  CUDA_CALL(hipFree(x_d));
  CUDA_CALL(hipFree(y_d));
  CUDA_CALL(hipFree(mu_x_d));
  CUDA_CALL(hipFree(mu_y_d));
  CUDA_CALL(hipFree(group_d));
  CUDA_CALL(hipFree(nx_d));
  CUDA_CALL(hipFree(ny_d));
  CUDA_CALL(hipFree(sum_x_d));
  CUDA_CALL(hipFree(sum_y_d));
  CUDA_CALL(hipFree(dst_d));

  return 0;
}

void read_data(float **x, float **y, float **mu_x, float **mu_y, int *n, int *k,char* arg){
  FILE *fp;
  char buf[64];

  *n = 0;
  fp = fopen(arg, "r");

  while(fgets(buf, 64, fp) != NULL){
    *n += 1;
    *x = (float*) realloc(*x, (*n)*sizeof(float));
    *y = (float*) realloc(*y, (*n)*sizeof(float));
    std::istringstream line_stream(buf);
    float x1,y1;
    line_stream >> x1 >> y1;
    (*x)[*n - 1] = x1;
    (*y)[*n - 1] = y1;
  }
  fclose(fp);

  
  *k = 0;
  fp = fopen("input/initCoord.txt", "r");
  while(fgets(buf, 64, fp) != NULL){
    *k += 1;
    *mu_x = (float*) realloc(*mu_x, (*k)*sizeof(float));
    *mu_y = (float*) realloc(*mu_y, (*k)*sizeof(float));
    std::istringstream line_stream(buf);
    float x1,y1;
    line_stream >> x1 >> y1;
    (*mu_x)[*k - 1] = x1;
    (*mu_y)[*k - 1] = x1;
  }
  fclose(fp);
}


void print_results(int *group, float *mu_x, float *mu_y, int n, int k,char* arg){
  FILE *fp;
  int i;
  std::string str(arg),str1,str2;
  str = "output/cuda/" + str;

   str1 = str + "_group_members.txt";
  fp = fopen(str1.c_str(), "w");
  for(i = 0; i < n; ++i){
    fprintf(fp, "%d\n", group[i]);
  }
  fclose(fp);
  
  str2 = str + "_centroids.txt";
  fp = fopen(str2.c_str(), "w");
  for(i = 0; i < k; ++i){
    fprintf(fp, "%0.6f %0.6f\n", mu_x[i], mu_y[i]);
  }
  fclose(fp);

  fp = fopen("CUDAtimes.txt", "a");
    fprintf(fp, "%0.6f\n", gpu_time_used);
fclose(fp);
}
